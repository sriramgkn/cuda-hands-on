
#include <hip/hip_runtime.h>
#include <stdio.h>

//a simple kernel that squares every element in an array
__global__ void square(float * d_in, float * d_out) {
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f;
}

//cpu program flow
int main(int argc, char ** argv) {
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    //generate input array, declare output array on cpu (host)
    float h_in[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    //declare gpu memory pointers, allocate gpu memory (device)
    float * d_in;
    float * d_out;
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    //transfer input from host to device
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);
    //launch kernel on device
    square<<<1, ARRAY_SIZE>>>(d_in, d_out);
    //transfer output from device to host
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    //print output
    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%f", h_out[i]);
        printf(((i%4) != 3) ? "\t" : "\n");
    }

    //free gpu memory
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}